#include <iostream>
#include <thread>
#include <atomic>
#include <chrono>
#include <hip/hip_runtime.h>

#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

constexpr size_t BLOCK_SIZE = 128*1024; // 128KB
constexpr size_t BUFFER_SIZE = 4; // 4 times of BLOCK_SIZE
#define WRITE_FREQ 1
#define READ_FREQ 0
#define DATA_SIZE 874*BLOCK_SIZE

#define WIDTH 1944
#define HEIGHT 1472
#define NUM_FRAME 20


std::atomic<size_t> write_ptr(0);
std::atomic<size_t> read_ptr(0);

char* ring_buffer = new char[BUFFER_SIZE * BLOCK_SIZE];
char* gpu_data_mem_head;
char* gpu_data_mem;
char* cpu_data_mem_head = new char[DATA_SIZE];
char* cpu_data_mem;

std::atomic <bool> producer_running(false);
std::atomic <bool> consumer_running(false);

void Producer() {
    while(producer_running) {
        if ((write_ptr + 1) % BUFFER_SIZE != read_ptr.load(std::memory_order_acquire)) {
            hipMemcpy(&ring_buffer[write_ptr * BLOCK_SIZE], gpu_data_mem, BLOCK_SIZE, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            if (gpu_data_mem == gpu_data_mem_head + DATA_SIZE - BLOCK_SIZE) {
                producer_running = false;
            } else {
                gpu_data_mem += BLOCK_SIZE;
            }
            write_ptr.store((write_ptr + 1) % BUFFER_SIZE, std::memory_order_release);
            std::this_thread::sleep_for(std::chrono::milliseconds(WRITE_FREQ));
        }
    }
}

void Consumer() {
    while(consumer_running) {
        if (read_ptr.load(std::memory_order_acquire) != write_ptr) {
            memcpy(cpu_data_mem, &ring_buffer[read_ptr * BLOCK_SIZE], BLOCK_SIZE);
            if (cpu_data_mem == cpu_data_mem_head + DATA_SIZE - BLOCK_SIZE) {
                consumer_running = false;
            } else {
                cpu_data_mem = cpu_data_mem + BLOCK_SIZE;
            }
            read_ptr.store((read_ptr + 1) % BUFFER_SIZE, std::memory_order_release);
            std::this_thread::sleep_for(std::chrono::milliseconds(READ_FREQ));
        }
    }
}

char* LoadImage(std::string image_folder) {
  int width = WIDTH;
  int height = HEIGHT;
  int num_frame = NUM_FRAME;


}
int main() {
    char* cpu_data = new char[DATA_SIZE];
    for (int i = 0; i < DATA_SIZE; ++i) {
        cpu_data[i] = i%256;
    }

    hipMalloc((void**)&gpu_data_mem_head, DATA_SIZE * sizeof(char));
    std::cout << "gpu_buffer size: " << DATA_SIZE * sizeof(char) << std::endl;

    hipMemcpy(gpu_data_mem_head, cpu_data, DATA_SIZE, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    gpu_data_mem = gpu_data_mem_head;
    
    cpu_data_mem = cpu_data_mem_head;
    producer_running = true;
    consumer_running = true;
    std::thread producer(Producer);
    std::thread consumer(Consumer);

    producer.join();
    consumer.join();

    hipDeviceSynchronize();
    std::cout << "Data transfer is done" << std::endl;

    for (int i = 0; i < DATA_SIZE; ++i) {
        if (cpu_data_mem_head[i] != cpu_data[i]) {
            printf("cpu_data_mem_head[%d]: %p  %d %d\n", i, cpu_data_mem_head+i, *(cpu_data_mem_head+i), *(cpu_data+i));    
            break;
        }   
    }
    std::cout << "Data transfer is correct" << std::endl;
    

    // Deallocate memory
    delete[] cpu_data;
    delete[] cpu_data_mem_head;
    delete[] ring_buffer;
    hipFree(gpu_data_mem_head);

    return 0;
}
